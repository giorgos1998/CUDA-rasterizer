#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "gpu_containers.h"

// Creates a new (0,0) point.
__host__ __device__ GPUPoint::GPUPoint()
{
    this->x = 0;
    this->y = 0;
}

/**
 * @brief Creates a new point with given coordinates.
 *
 * @param x The X coordinate.
 * @param y The Y coordinate.
 */
__host__ __device__ GPUPoint::GPUPoint(float x, float y)
{
    this->x = x;
    this->y = y;
}

// Copy constructor
__host__ __device__ GPUPoint::GPUPoint(GPUPoint &that)
{
    this->x = that.x;
    this->y = that.y;
}

// Destructor
__host__ __device__ GPUPoint::~GPUPoint()
{
    // printf("Deleting point\n");
}

// Prints the coordinates of the point.
__host__ __device__ void GPUPoint::print()
{
    printf("(%f, %f)\n", x, y);
}

/**
 * @brief Creates a new Polygon that can be used from host and device.
 *
 * @param size The number of points (vertices) the polygon has.
 * @param points The array of points.
 */
__host__ __device__ GPUPolygon::GPUPolygon(int size, GPUPoint points[])
{
    this->size = size;
    this->hilbertPoints = points;
    this->points = new GPUPoint[size];
}

// Copy constructor
__host__ __device__ GPUPolygon::GPUPolygon(GPUPolygon &that)
{
    // printf("In copy constructor\n");

    this->size = that.size;
    this->points = new GPUPoint[this->size];

    for (int i = 0; i < size; i++)
    {
        this->points[i] = that.points[i];
    }
}

// Copy assignment operator
__host__ __device__ GPUPolygon &GPUPolygon::operator=(const GPUPolygon &that)
{
    // printf("In copy assignment operator\n");
    if (this != &that)
    {
        // Using assert to work both on host and device.
        // Stop execution if the polygons don't have the same size.
        assert(this->size == that.size);

        this->size = that.size;
        this->points = new GPUPoint[this->size];

        for (int i = 0; i < size; i++)
        {
            this->points[i] = that.points[i];
        }
    }
    return *this;
}

// Destructor
__host__ __device__ GPUPolygon::~GPUPolygon()
{
    // printf("Deleting polygon\n");
}

// Prints the polygon points.
__host__ __device__ void GPUPolygon::print()
{
    printf("----- Polygon -----\n");
    printf("Hilbert min: ");
    hMin.print();
    printf("Hilbert max: ");
    hMax.print();
    printf("Points:\n");
    for (int i = 0; i < size; i++)
    {
        points[i].print();
    }
}

// Prints polygon's rasterization matrix.
__host__ __device__ void GPUPolygon::printMatrix()
{
    printf("Rasterization matrix:\n");
    printf("Size (WxH): %dx%d\n", mbrWidth, mbrHeight);
    printf("   ");
    for (int i = 0; i < mbrWidth; i++)
    {
        printf("%d ", i%10);
    }
    printf("\n");
    for (int i = 0; i < mbrHeight; i++)
    {
        printf("%2d ", i);
        for (int j = 0; j < mbrWidth; j++)
        {
            if (matrix[(i * mbrWidth) + j] == 3)
            {
                printf("? ");
            }
            else if (matrix[(i * mbrWidth) + j] == 0)
            {
                // printf(" ");
                printf("\u00B7 ");
            }
            else if (matrix[(i * mbrWidth) + j] == 1)
            {
                printf("\U000025A0 ");
                // printf("\U000025CF ");
            }
            else if (matrix[(i * mbrWidth) + j] == 2)
            {
                printf("\U000025A3 ");
            }
            else
            {
                printf("%d ", matrix[(i * mbrWidth) + j]);
            }
        }
        printf("\n");
    }
}

// Creates a new empty stack
__host__ __device__ GPUStack::GPUStack()
{
    size = 0;
    lastItem = NULL;
}

/**
 * @brief Add an item in the stack.
 * 
 * @param x The X coordinate of the point to add.
 * @param y The Y coordinate of the point to add.
 */
__host__ __device__ void GPUStack::push(int x, int y)
{
    GPUStackItem *item = new GPUStackItem;
    item->point = GPUPoint(x, y);

    item->prevItem = lastItem;
    lastItem = item;
    size++;

    // printf("Added ");
    // item->point.print();
    // printf("Current item %p\n", item);
    // printf("Previous item %p\n", item->prevItem);
}

// Removes and returns last item from the stack.
__host__ __device__ GPUPoint GPUStack::pop()
{
    // Stop execution if trying to pop from empty stack
    assert(size > 0);

    // Copy poped item
    GPUStackItem poped;
    poped.point = GPUPoint(lastItem->point);
    poped.prevItem = lastItem->prevItem;

    // Delete poped item
    lastItem->point.~GPUPoint();
    delete lastItem;

    // Change stack pointer & size
    lastItem = poped.prevItem;
    size--;

    // printf("Removed ");
    // poped.point.print();

    return poped.point;
}

// Destructor
__host__ __device__ GPUStack::~GPUStack()
{
    while (size > 0)
    {
        this->pop();
    }
}

__host__ __device__ void GPUStack::print()
{
    GPUStackItem *currItem = lastItem;

    printf("Current stack items:\n");
    for (int i = size; i > 0; i--)
    {
        printf("%2d: ", i);
        currItem->point.print();
        currItem = currItem->prevItem;
    }
}