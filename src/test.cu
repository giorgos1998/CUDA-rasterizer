#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu_containers.h"

__global__ void printPolygon(GPUPolygon &poly)
{
    printf("Hello from the GPU!\n");
    poly.print();
}

int main()
{
    GPUPoint points[3];

    for (int i = 0; i < 3; i++)
    {
        points[i] = GPUPoint(i + 1, i + 1);
    }

    GPUPolygon poly1 = GPUPolygon(3, points);
    // poly1.print();

    // Copy polygon to device
    GPUPolygon *polyD;
    hipMalloc((void **)&polyD, sizeof(GPUPolygon));
    hipMemcpy(polyD, &poly1, sizeof(GPUPolygon), hipMemcpyHostToDevice);

    // Copy points to device
    GPUPoint *pointsD;
    hipMalloc((void **)&pointsD, poly1.size * sizeof(GPUPoint));
    hipMemcpy(pointsD, poly1.points, poly1.size * sizeof(GPUPoint), hipMemcpyHostToDevice);

    // Set device polygon points pointer to copied points
    hipMemcpy(&(polyD->points), &pointsD, sizeof(GPUPoint *), hipMemcpyHostToDevice);

    printPolygon<<<1, 1>>>(*polyD);
    hipDeviceSynchronize();

    hipFree(polyD);
    hipFree(pointsD);

    return 0;
}